#include "hip/hip_runtime.h"
#include "tensor/native/kernels.cuh"
#include "tensor/tensor.h"

template<typename scalar_t>
__global__ void matmul_naive_kernel(const scalar_t* a, const scalar_t* b, scalar_t* c, int32_t m, int32_t k, int32_t n){
    // MxK, KxN = MxN
    int x = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y = (blockDim.y * blockIdx.y) + threadIdx.y;

    if(x < n && y < m){
        scalar_t sum = scalar_t(0);
        for(int i = 0; i < k; ++i){
            sum  += a[y*k + i] * b[i*n + x];
        }
        c[y*n + x] = sum;
    }
}

template<typename scalar_t>
__global__ void matmul_naive_gradA_kernel(const scalar_t* grad_out, const scalar_t* b, scalar_t* da, int32_t m, int32_t k, int32_t n){
        int x = (blockDim.x * blockIdx.x) + threadIdx.x;
        int y = (blockDim.y * blockIdx.y) + threadIdx.y;

        // grad_out -> (m,n)
        // a -> (m,k)
        // b -> (k,n)
        // da = grad_out @ b.scalar_t -> (m,n) @ (n,k) = (m,k)
        if(x >= k || y >= m) return;

        scalar_t sum = scalar_t(0);
        for(int i = 0; i < n; ++i){
            sum += grad_out[y*n + i]*b[x*n + i];
        }

        da[y*k + x] = sum;
}

template<typename scalar_t>
__global__ void matmul_naive_gradB_kernel(
    const scalar_t* grad_out, const scalar_t* a, scalar_t* db, int32_t m, int32_t k, int32_t n){
        int x = (blockDim.x * blockIdx.x) + threadIdx.x;
        int y = (blockDim.y * blockIdx.y) + threadIdx.y;

        // grad_out -> (m,n)
        // a -> (m,k)
        // b -> (k,n)
        // db = a.scalar_t @ grad_out = (k,m) @ (m,n) = (k,n)
        if(x >= n || y >= k) return;

        scalar_t sum = scalar_t(0);
        for(int i = 0; i < m; ++i){
            sum += a[i*k + y]*grad_out[i*n + x];
        }

        db[y*n + x] = sum;
}

// TODO: Macro this for custom types.
template __global__ void matmul_naive_kernel<float>( const float*, const float*, float*, int32_t, int32_t, int32_t);
template __global__ void matmul_naive_gradA_kernel<float>( const float*, const float*, float*, int32_t, int32_t, int32_t);
template __global__ void matmul_naive_gradB_kernel<float>( const float*, const float*, float*, int32_t, int32_t, int32_t);