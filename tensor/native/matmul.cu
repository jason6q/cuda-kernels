#include "hip/hip_runtime.h"
#include <vector>

#include "core/data_ptr.h"
#include "core/device.h"
#include "tensor/macros.h"
#include "tensor/ops.h"
#include "tensor/tensor.h"
#include "tensor/native/kernels.cuh"

namespace jqTen{
    Tensor matmul_tile_cuda(const Tensor& a, const Tensor& b){
        JQ_ASSERT(a.device() == core::Device::CUDA, "Tensor a device not CUDA");
        JQ_ASSERT(b.device() == core::Device::CUDA, "Tensor b device not CUDA");

        std::vector<int32_t> a_shape = a.shape();
        std::vector<int32_t> b_shape = a.shape();

        int32_t m = a_shape.back();                 // (..., m, n)
        int32_t n = b_shape[b_shape.size()-1];      // (... n, k)
        int32_t k = b_shape.back();

        Tensor c = empty({m,k});
        c.to(core::Device::CUDA);

        const float* a_buf = static_cast<const float*>(a.data());
        const float* b_buf = static_cast<const float*>(b.data());
        float* c_buf = static_cast<float*>(c.data());

        // Kernel Launch
        int32_t GRID_SIZE = 16;
        int32_t BLOCK_SIZE = 16;
        int32_t TILE_SIZE = BLOCK_SIZE;

        dim3 gridDim(16,16);
        dim3 blockDim(16,16); // Will be tile size

        // SHMEM Size
        int32_t shmem_size = 2*TILE_SIZE*TILE_SIZE*sizeof(float);
        matmul_tile_kernel<float><<<gridDim, blockDim, shmem_size>>>(a_buf, b_buf, c_buf, m, n, k, TILE_SIZE);
    }

    Tensor matmul_naive_cuda(const Tensor& a, const Tensor& b){
        JQ_ASSERT(a.device() == core::Device::CUDA, "Tensor a device not CUDA");
        JQ_ASSERT(b.device() == core::Device::CUDA, "Tensor b device not CUDA");

        std::vector<int32_t> a_shape = a.shape();
        std::vector<int32_t> b_shape = b.shape();

        int32_t m = a_shape.back();
        int32_t n = b_shape[b_shape.size()-1];
        int32_t k = b_shape.back();

        // Calculate c output dim
        // Take a = {..., M, N}, b = {..., N, K}, c = {..., M, K}
        Tensor c = empty({m,k});
        c.to(core::Device::CUDA);

        // TODO: Template the types here.
        // Allow this section of the code to handle different types.
        const float* a_buf = static_cast<const float*>(a.data());
        const float* b_buf = static_cast<const float*>(b.data());
        float* c_buf = static_cast<float*>(c.data());

        // Kernel launch
        dim3 blockDim = dim3(16,16);
        dim3 gridDim = dim3(16,16);

        // TODO: Specify templated scalar_t instead of just float.
        //       May need to make a similar macro like in torch.
        matmul_naive_kernel<float><<<gridDim, blockDim, 0>>>(a_buf, b_buf, c_buf, m,k,n);

        return c;
    }
}